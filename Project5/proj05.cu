#include "hip/hip_runtime.h"
// Monte Carlo simulation of castle-bombardment:

// system includes
#include <stdio.h>
#include <assert.h>
#include <malloc.h>
#include <math.h>
#include <stdlib.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include "helper_functions.h"
#include "hip/hip_runtime_api.h"

// setting the number of trials in the monte carlo simulation:
#ifndef NUMTRIALS
#define NUMTRIALS ( 8*1024*1024 )
#endif

// number of threads per block:
#ifndef BLOCKSIZE
#define BLOCKSIZE 64
#endif

// number of blocks:
#define NUMBLOCKS ( NUMTRIALS / BLOCKSIZE )

// better to define these here so that the rand() calls don't get into the thread timing:
float hvs[NUMTRIALS];
float hths[NUMTRIALS];
float hgs[NUMTRIALS];
float hhs[NUMTRIALS];
float hds[NUMTRIALS];
int hhits[NUMTRIALS];

// ranges for the random numbers:
const float GMIN = 20.0; // ground distance in meters
const float GMAX = 30.0; // ground distance in meters
const float HMIN = 10.0; // cliff height in meters
const float HMAX = 20.0; // cliff height in meters
const float DMIN = 10.0; // distance to castle in meters
const float DMAX = 20.0; // distance to castle in meters
const float VMIN = 10.0; // initial cannonball velocity in meters / sec
const float VMAX = 30.0; // initial cannonball velocity in meters / sec
const float THMIN = 70.0; // cannonball launch angle in degrees
const float THMAX = 80.0; // cannonball launch angle in degrees

// constants:
const float GRAVITY = -9.8; // acceleration due to gravity in meters / sec^2
const float TOL = 5.0; // tolerance in cannonball hitting the castle in meters

// function prototypes:
void CudaCheckError( );
float Ranf( float, float );
void TimeOfDaySeed( );

// degrees-to-radians -- callable from the device:
__device__
float
Radians( float d )
{
    return (M_PI/180.f) * d;
}

// the kernel:
__global__
void
MonteCarlo( float *dvs, float *dths, float *dgs, float *dhs, float *dds, int *dhits )
{
    unsigned int gid = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the thread is within bounds
    if (gid >= NUMTRIALS)
        return;

    // Randomize everything:
    float v   = dvs[gid];
    float thr = Radians( dths[gid] );
    float vx  = v * cos(thr);
    float vy  = v * sin(thr);
    float g   = dgs[gid];
    float h   = dhs[gid];
    float d   = dds[gid];

    dhits[gid] = 0;

    // See if the ball reaches the cliff:
    float t = -vy / ( 0.5 * GRAVITY );
    float x = vx * t;
    if ( x >= g ) // Corrected: proceed only if ball reaches or passes cliff
    {
        // See if the ball clears the vertical cliff face:
        t = g / vx;
        float y = vy * t + 0.5 * GRAVITY * t * t;
        if ( y >= h ) // Corrected: proceed only if ball clears cliff face
        {
            // The ball hits the upper deck:
            float a = 0.5 * GRAVITY;
            float b = vy;
            float c = -h;
            float disc = b * b - 4.f * a * c; // quadratic formula discriminant

            // Successfully hits the ground above the cliff:
            // Get the intersection:
            disc = sqrtf( disc );
            float t1 = (-b + disc ) / ( 2.f * a ); // time to intersect high ground
            float t2 = (-b - disc ) / ( 2.f * a ); // time to intersect high ground
            float tmax = t1;
            if ( t2 > tmax )
                tmax = t2; // only care about the second intersection

            // How far does the ball land horizontally from the edge of the cliff?
            float upperDist = vx * tmax - g;

            // See if the ball hits the castle:
            if ( fabs( upperDist - d ) <= TOL )
            {
                dhits[gid] = 1;
            }
        } // if ball clears the cliff face
    } // if ball gets as far as the cliff face
}

// main program:
int
main( int argc, char* argv[ ] )
{
    TimeOfDaySeed( );

    // fill the random-value arrays:
    for( int n = 0; n < NUMTRIALS; n++ )
    {
        hvs[n]  = Ranf( VMIN, VMAX );
        hths[n] = Ranf( THMIN, THMAX );
        hgs[n]  = Ranf( GMIN, GMAX );
        hhs[n]  = Ranf( HMIN, HMAX );
        hds[n]  = Ranf( DMIN, DMAX );
    }

    // allocate device memory:
    float *dvs, *dths, *dgs, *dhs, *dds;
    int *dhits;

    hipMalloc( &dvs, NUMTRIALS * sizeof(float) );
    hipMalloc( &dths, NUMTRIALS * sizeof(float) );
    hipMalloc( &dgs, NUMTRIALS * sizeof(float) );
    hipMalloc( &dhs, NUMTRIALS * sizeof(float) );
    hipMalloc( &dds, NUMTRIALS * sizeof(float) );
    hipMalloc( &dhits, NUMTRIALS * sizeof(int) );
    CudaCheckError( );

    // copy host memory to the device:
    hipMemcpy( dvs, hvs, NUMTRIALS * sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy( dths, hths, NUMTRIALS * sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy( dgs, hgs, NUMTRIALS * sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy( dhs, hhs, NUMTRIALS * sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy( dds, hds, NUMTRIALS * sizeof(float), hipMemcpyHostToDevice );
    CudaCheckError( );

    // setup the execution parameters:
    dim3 grid( NUMBLOCKS, 1, 1 );
    dim3 threads( BLOCKSIZE, 1, 1 );

    // allocate cuda events that we'll use for timing:
    hipEvent_t start, stop;
    hipEventCreate( &start );
    hipEventCreate( &stop );
    CudaCheckError( );

    // let the gpu go quiet:
    hipDeviceSynchronize( );

    // record the start event:
    hipEventRecord( start, NULL );
    CudaCheckError( );

    // execute the kernel:
    MonteCarlo<<< grid, threads >>>( dvs, dths, dgs, dhs, dds, dhits );

    // record the stop event:
    hipEventRecord( stop, NULL );
    CudaCheckError( );

    // wait for the stop event to complete:
    hipDeviceSynchronize( );
    hipEventSynchronize( stop );
    CudaCheckError( );

    float msecTotal = 0.0f;
    hipEventElapsedTime( &msecTotal, start, stop );
    CudaCheckError( );

    // compute and print the performance
    double secondsTotal = 0.001 * (double)msecTotal;
    double trialsPerSecond = (float)NUMTRIALS / secondsTotal;
    double megaTrialsPerSecond = trialsPerSecond / 1000000.;

    // copy result from the device to the host:
    hipMemcpy( hhits, dhits, NUMTRIALS * sizeof(int), hipMemcpyDeviceToHost );
    CudaCheckError( );

    // compute the sum:
    int numHits = 0;
    for(int i = 0; i < NUMTRIALS; i++ )
    {
        numHits += hhits[i];
    }

    // compute the probability (only in non-CSV mode):
    float probability = 100.f * (float)numHits / (float)NUMTRIALS;

//#define CSV


#ifdef CSV
    fprintf( stderr, "%10d , %5d , %8.2lf, %6.3f\n", NUMTRIALS, BLOCKSIZE, megaTrialsPerSecond, probability );
#else
    fprintf( stderr, "Trials = %10d, BlockSize = %5d, MegaTrials/Second = %8.2lf, Probability=%6.3f%%\n",
        NUMTRIALS, BLOCKSIZE, megaTrialsPerSecond, probability );
#endif

    // clean up device memory:
    hipFree( dvs );
    hipFree( dths );
    hipFree( dgs );
    hipFree( dhs );
    hipFree( dds );
    hipFree( dhits );
    CudaCheckError( );

    // done:
    return 0;
}

void
CudaCheckError( )
{
    hipError_t e = hipGetLastError( );
    if( e != hipSuccess )
    {
        fprintf( stderr, "CUDA failure %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(e) );
    }
}

float
Ranf( float low, float high )
{
    float r = (float) rand();               // 0 - RAND_MAX
    float t = r / (float) RAND_MAX;       // 0. - 1.
    return low + t * ( high - low );
}

void
TimeOfDaySeed( )
{
    time_t now;
    time( &now );
    struct tm n = *localtime(&now);

    struct tm jan01 = *localtime(&now);
    jan01.tm_mon = 0;
    jan01.tm_mday = 1;
    jan01.tm_hour = 0;
    jan01.tm_min = 0;
    jan01.tm_sec = 0;

    double seconds = difftime( now, mktime(&jan01) );
    unsigned int seed = (unsigned int)( 1000.*seconds );    // milliseconds
    srand( seed );
}
